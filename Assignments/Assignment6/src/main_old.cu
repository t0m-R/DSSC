
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 8192 // Number of rows/columns of the matrix.
#define TILE_DIM 32
#define SIZE N*N // Total size of a matrix.


// Compares two matrices element by element.
int isTransposed (const double* a, const double* b, const int dim) {
  for (int i = 0; i < dim; i++) {
    for (int j = 0; j < dim; j++) {
      if(b[j + i*dim] != a[i + j*dim]) return 0;
    }
  }
  return 1;
}



// Gpu naive transposition.
__global__ void gpuNaiveTrans (double* a, double* b, const int size,
                               const int brows) {

  int col = blockIdx.x * TILE_DIM + threadIdx.x;
  int row = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for(int i = 0; i < TILE_DIM; i += brows) {
    b[col * width + (row + i)] = a[(row + i) * width + col];
  }
}



// Gpu optimised transposition.
__global__ void gpuOptTrans (double* a, double* b, const int size,
                             const int brows) {

  // Buffer on the shared memory.
  __shared__ double tmp[TILE_DIM][TILE_DIM];

  int col = blockIdx.x * TILE_DIM + threadIdx.x;
  int row = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  // Transposition on buffer.
  for(int i = 0; i < TILE_DIM; i += brows) {
    tmp[threadIdx.y + i][threadIdx.x] = a[(row + i) * width + col];
  }

  __syncthreads();

  col = blockIdx.y * TILE_DIM + threadIdx.x;
  row = blockIdx.x * TILE_DIM + threadIdx.y;

  // Writing to output.
  for(int i = 0; i < TILE_DIM; i += brows) {
    b[col + width * (row + i)] = tmp[threadIdx.x][threadIdx.y + i];
  }
}



void matrixFill (double* a, const int dim) {
  for(int i = 0; i < dim; i++) {
    a[i] = (double) i;
  }
}



int main(int argc, char* argv[]) {

  double* hostInput, * hostOutput;
  double* devInput, * devOutput;

  // Allocate host memory.
  hostInput = (double* )malloc(SIZE * sizeof(double));
  hostOutput = (double* )malloc(SIZE * sizeof(double));

  // Allocate device memory.
  hipMalloc((void**)&devInput, SIZE * sizeof(double));
  hipMalloc((void**)&devOutput, SIZE * sizeof(double));

  // Dimensions.
  dim3 grid, block;
  block.x = TILE_DIM;
  block.y = atoi(argv[1])/TILE_DIM; // Threads-per-block readed as input.
  grid.x = N / TILE_DIM;
  grid.y = N / TILE_DIM;

  printf("\n--------------------------\n");
  printf("--------------------------\n\n");

  printf("Threads per block = %d\n\n", atoi(argv[1]));


  /// NAIVE TRANSPOSE //

  // Initialise matricx.
  matrixFill(hostInput, SIZE);

  // Copy input to device.
  hipMemcpy(devInput, hostInput, SIZE * sizeof(double),
             hipMemcpyHostToDevice);

  // Timing.
  float elapsedTime = 0.0;
  hipEvent_t tStart, tEnd;
  hipEventCreate(&tStart);
  hipEventCreate(&tEnd);

  hipEventRecord(tStart);
    gpuNaiveTrans<<< grid, block >>>(devInput, devOutput, N, block.y);
  hipEventRecord(tEnd);

  hipEventSynchronize(tEnd);
  hipEventElapsedTime(&elapsedTime, tStart, tEnd);
  printf("NAIVE TRANSPOSE: Elapsed time: %fms\n", elapsedTime);
  printf("Bandwidth: %f GB/s\n",
         2 * SIZE * sizeof(double) / elapsedTime / 1000000);

  // Copy output to host.
  hipMemcpy(hostOutput, devOutput, SIZE * sizeof(double),
             hipMemcpyDeviceToHost);
  
  printf("Is the tranposition correct? %s\n",
         isTransposed(hostOutput, hostInput, N) ? "CORRECT" : "ERROR!" );



  /// OPTIMISED TRANSPOSE //

  printf("\n\n");

  // Initialise matricx.
  matrixFill(hostInput, SIZE);

  // Copy input to device.
  hipMemcpy(devInput, hostInput, SIZE * sizeof(double),
             hipMemcpyHostToDevice);

  // Timing.
  elapsedTime = 0.0;

  hipEventRecord(tStart);
    gpuOptTrans<<< grid, block >>>(devInput, devOutput, N, block.y);
  hipEventRecord(tEnd);

  hipEventSynchronize(tEnd);
  hipEventElapsedTime(&elapsedTime, tStart, tEnd);
  printf("OPTIMISED TRANSPOSE: Elapsed time: %fms\n", elapsedTime);
  printf("Bandwidth: %f GB/s\n",
         2 * SIZE * sizeof(double) / elapsedTime / 1000000);

  // Copy output to host.
  hipMemcpy(hostOutput, devOutput, SIZE * sizeof(double),
             hipMemcpyDeviceToHost);
  
  printf("Is the tranposition correct? %s\n",
         isTransposed(hostOutput, hostInput, N) ? "CORRECT" : "ERROR!" );



  // Freeing resources.
  free(hostInput);
  free(hostOutput);
  hipFree(devInput);
  hipFree(devOutput);
  hipEventDestroy(tStart);
  hipEventDestroy(tEnd);

}
