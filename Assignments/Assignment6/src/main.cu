
#include <hip/hip_runtime.h>
#include <stdio.h>


#define N 8192
#define TILE 32
#define SIZE N*N


__global__ void transpose_gpu(double *b, const double *a, const int size)
{
    int x = blockIdx.x * TILE + threadIdx.x;
    int y = blockIdx.y * TILE + threadIdx.y;
    int width = gridDim.x * TILE;

    for (int i = 0; i < TILE; i+= size)
        b[x*width + (y + i)] = a[(y + i) * width + x];
}


__global__ void transpose_gpu_opt(double *b, const double *a, const int size)
{
    __shared__ double tmp[TILE][TILE];

    int x = blockIdx.x * TILE + threadIdx.x;
    int y = blockIdx.y * TILE + threadIdx.y;
    int id_a = x + (y) * N;

    x = blockIdx.y * TILE + threadIdx.x;
    y = blockIdx.x * TILE + threadIdx.y;
    int id_b = x + (y) * N;

    for (int i = 0; i < TILE; i+=size) {
        tmp[threadIdx.y + i][threadIdx.x] = a[id_a + i * N];
    }
    __syncthreads();

    for (int  i = 0; i < TILE; i+=size) {
        b[id_b + i * N] = tmp[threadIdx.x][threadIdx.y + i];
    }
}


void fill_matrix (double* a, const int dim) {
    for(int i = 0; i < dim; i++) {
        a[i] = (double) i;
    }
}


int main(int argc, char const *argv[]) {

    dim3 grid, block;
    block.x = TILE;
    block.y = atoi(argv[1]);
    grid.x = N / TILE;
    grid.y = N / TILE;

    double* host_input, * host_output;
    double* gpu_input, * gpu_output;

    host_input = (double* )malloc(SIZE * sizeof(double));
    host_output = (double* )malloc(SIZE * sizeof(double));

    hipMalloc((void**)&gpu_input, SIZE * sizeof(double));
    hipMalloc((void**)&gpu_output, SIZE * sizeof(double));

    char sep[] = "---------------------------------------------\n";
    printf("%sTHREADS x BLOCK: %d\n%s", sep, atoi(argv[1]), sep);


    //TRANSPOSE.
    fill_matrix(host_input, SIZE);

    hipMemcpy(gpu_input, host_input, SIZE, hipMemcpyHostToDevice);

    float total_time, total_time_opt = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    transpose_gpu<<<grid,block>>>(gpu_input, gpu_output, block.y);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&total_time, start, stop);

    double bandwidth = 2. * (double)SIZE * sizeof(double) / total_time / 1e6;
    printf("TRANSPOSE:\t\t %fms\n", total_time);
    printf("BANDWIDTH:\t\t %fms\n%s", bandwidth, sep);

    hipMemcpy(host_output, gpu_output, SIZE, hipMemcpyDeviceToHost);


    // TRANSPOSE OPT.
    fill_matrix(host_input, SIZE);

    hipMemcpy(gpu_input, host_input, SIZE, hipMemcpyHostToDevice);

    hipEventRecord(start,0);

    transpose_gpu_opt<<<grid, block>>>(gpu_input, gpu_output, block.y);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&total_time_opt, start, stop);

    double bandwidth_opt = 2. * (double)SIZE * sizeof(double) / total_time_opt / 1e6;
    printf("TRANSPOSE OPT:\t\t %fms\n", total_time_opt);
    printf("BANDWIDTH:\t\t %fms\n%s\n", bandwidth_opt, sep);

    hipMemcpy(host_output, gpu_output, SIZE, hipMemcpyDeviceToHost);


    free(host_input);
    free(host_output);
    hipFree(gpu_input);
    hipFree(gpu_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
